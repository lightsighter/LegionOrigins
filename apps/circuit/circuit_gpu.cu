
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "circuit.h"

__host__
void calc_new_currents_gpu(CircuitPiece *p,
                           GPU_Accessor wires,
                           GPU_Accessor pvt,
                           GPU_Accessor owned,
                           GPU_Accessor ghost)
{

}

__host__
void distribute_charge_gpu(CircuitPiece *p,
                           GPU_Accessor wires,
                           GPU_Accessor pvt,
                           GPU_Reducer owned,
                           GPU_Reducer ghost)
{

}

__host__
void update_voltages_gpu(CircuitPiece *p,
                         GPU_Accessor pvt,
                         GPU_Accessor owned)
{

}
