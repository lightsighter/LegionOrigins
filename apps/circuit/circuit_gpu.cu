#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "circuit.h"

#define CUDA_SAFE_CALL(expr)				\
	{						\
		hipError_t err = (expr);		\
		if (err != hipSuccess)			\
		{					\
			printf("Cuda error: %s\n", hipGetErrorString(err));	\
			assert(false);			\
		}					\
	}


class GPUAccumulateCharge {
public:
  typedef CircuitNode LHS;
  typedef float RHS;

  template<bool EXCLUSIVE>
  __device__ __forceinline__
  static void apply(LHS &lhs, RHS &rhs)
  {
    float *target = &(lhs.charge); 
    atomicAdd(target,rhs);
  }

  template<bool EXCLUSIVE>
  __device__ __forceinline__
  static void fold(RHS &rhs1, RHS rhs2)
  {
    float *target = &rhs1;
    atomicAdd(target,rhs2);
  }
};

__device__ __forceinline__
CircuitNode& get_node(GPU_Accessor pvt, GPU_Accessor owned, GPU_Accessor ghost, 
                      PointerLocation loc, ptr_t<CircuitNode> ptr)
{
  switch (loc)
  {
    case PRIVATE_PTR:
      //assert((pvt.first_elmt <= ptr.value) && (ptr.value <= pvt.last_elmt));
      return pvt.ref(ptr);
    case SHARED_PTR:
      //assert((owned.first_elmt <= ptr.value) && (ptr.value <= owned.last_elmt));
      return owned.ref(ptr);
    case GHOST_PTR:
      //assert((ghost.first_elmt <= ptr.value) && (ptr.value <= ghost.last_elmt));
      return ghost.ref(ptr);
    default:
      assert(false);
  }
  return pvt.ref(ptr);
}

__global__
void calc_new_currents_kernel(ptr_t<CircuitWire> first,
                              int num_wires,
                              GPU_Accessor wires,
                              GPU_Accessor pvt,
                              GPU_Accessor owned,
                              GPU_Accessor ghost,
                              int flag)
{
#ifndef DISABLE_MATH
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 

  if (tid < num_wires)
  {
    ptr_t<CircuitWire> local_ptr;
    local_ptr.value = first.value + tid;
    //if(tid == 0) printf("i am %d (w=%d) %p\n", tid, local_ptr.value, wires.array_base);
    //assert((wires.first_elmt <= local_ptr.value) && (local_ptr.value <= wires.last_elmt));
    CircuitWire &wire = wires.ref(local_ptr);
    //CircuitWire wire = wires.read(local_ptr);
    //if(//((local_ptr.value >= 300220) && (local_ptr.value <= 300229)) ||
       //((local_ptr.value >= 299710) && (local_ptr.value <= 299720)) ||
    //   (wire.in_loc < 0) || (wire.in_loc > 2) ||
    //   (wire.out_loc < 0) || (wire.out_loc > 2) ||
    //   (wire.in_ptr.value < ((local_ptr.value/10000)*2500)) ||
    //   (wire.in_ptr.value >= (((local_ptr.value/10000)+1)*2500)))
    //  printf("wire in[%d] = %d(%d) -> %d(%d)\n",
    //	     local_ptr.value, wire.in_ptr.value, wire.in_loc,
    //	     wire.out_ptr.value, wire.out_loc);

    //if(blockIdx.x == 17)
    //  printf("nodes[%d] = %d(%d) -> %d(%d)\n",
    //     tid, wire.in_ptr.value, wire.in_loc, wire.out_ptr.value, wire.out_loc);
#ifndef ALL_PRIVATE
    CircuitNode &in_node = get_node(pvt, owned, ghost, wire.in_loc, wire.in_ptr);
    CircuitNode &out_node = get_node(pvt, owned, ghost, wire.out_loc, wire.out_ptr);
#else
    CircuitNode &in_node = pvt.ref(wire.in_ptr);
    CircuitNode &out_node = pvt.ref(wire.out_ptr);
#endif

    // Solve RLC model iteratively
    float dt = DELTAT;
    const int steps = STEPS;
    float new_i[WIRE_SEGMENTS];
    float new_v[WIRE_SEGMENTS+1];
    for (int i = 0; i < WIRE_SEGMENTS; i++)
      new_i[i] = wire.current[i];
    for (int i = 0; i < (WIRE_SEGMENTS-1); i++)
      new_v[i+1] = wire.voltage[i];
    new_v[0] = in_node.voltage;
    new_v[WIRE_SEGMENTS] = out_node.voltage;

    for (int j = 0; j < steps; j++)
    {
      // first, figure out the new current from the voltage differential
      // and our inductance:
      // dV = R*I + L*I' ==> I = (dV - L*I')/R
      for (int i = 0; i < WIRE_SEGMENTS; i++)
      {
        new_i[i] = ((new_v[i+1] - new_v[i]) - 
                    (wire.inductance*(new_i[i] - wire.current[i])/dt)) / wire.resistance;
      }
      // Now update the inter-node voltages
      for (int i = 0; i < (WIRE_SEGMENTS-1); i++)
      {
        new_v[i+1] = wire.voltage[i] + dt*(new_i[i] - new_i[i+1]) / wire.capacitance;
      }
    }

    // Copy everything back
    for (int i = 0; i < WIRE_SEGMENTS; i++)
      wire.current[i] = new_i[i];
    for (int i = 0; i < (WIRE_SEGMENTS-1); i++)
      wire.voltage[i] = new_v[i+1];
    //wires.write(local_ptr, wire);

    //if(//((local_ptr.value >= 300220) && (local_ptr.value <= 300229)) ||
    //   //((local_ptr.value >= 299710) && (local_ptr.value <= 299720)) ||
    //   (wire.in_loc < 0) || (wire.in_loc > 2) ||
    //   (wire.out_loc < 0) || (wire.out_loc > 2) ||
    //   (wire.in_ptr.value < ((local_ptr.value/10000)*2500)) ||
    //   (wire.in_ptr.value >= (((local_ptr.value/10000)+1)*2500)))
    //  printf("wire out[%d] = %d(%d) -> %d(%d)\n",
    //	     local_ptr.value, wire.in_ptr.value, wire.in_loc,
    //	     wire.out_ptr.value, wire.out_loc);
  }
#endif
}

__global__
void sanity_check_positions(ptr_t<CircuitWire> first,
                            int num_wires,
                            GPU_Accessor wires)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 

  if (tid < num_wires)
  {
    ptr_t<CircuitWire> local_ptr;
    local_ptr.value = first.value + tid;
    //if(tid == 0) printf("i am %d (w=%d) %p\n", tid, local_ptr.value, wires.array_base);
    CircuitWire &wire = wires.ref(local_ptr);
    //CircuitWire wire = wires.read(local_ptr);
   
    assert((wire.in_loc == PRIVATE_PTR) || (wire.in_loc == SHARED_PTR) || (wire.in_loc == GHOST_PTR));
    assert((wire.out_loc == PRIVATE_PTR) || (wire.out_loc == SHARED_PTR) || (wire.out_loc == GHOST_PTR));

    if(//((local_ptr.value >= 300220) && (local_ptr.value <= 300229)) ||
       //((local_ptr.value >= 299710) && (local_ptr.value <= 299720)) ||
       (wire.in_loc < 0) || (wire.in_loc > 2) ||
       (wire.out_loc < 0) || (wire.out_loc > 2) ||
       (wire.in_ptr.value < ((local_ptr.value/10000)*2500)) ||
       (wire.in_ptr.value >= (((local_ptr.value/10000)+1)*2500)))
      printf("wire check[%d] = %d(%d) -> %d(%d)\n",
	     local_ptr.value, wire.in_ptr.value, wire.in_loc,
	     wire.out_ptr.value, wire.out_loc);

    if (local_ptr.value == 300225)
    {
      printf("Wire %d has pointers %d %d at locations %d %d\n",local_ptr.value, wire.in_ptr.value, wire.out_ptr.value, wire.in_loc, wire.out_loc);
    }
  }
}

__host__
void calc_new_currents_gpu(CircuitPiece *p,
                           GPU_Accessor wires,
                           GPU_Accessor pvt,
                           GPU_Accessor owned,
                           GPU_Accessor ghost,
                           int flag)
{
  RegionRuntime::LowLevel::DetailedTimer::ScopedPush sp(TIME_SYSTEM+1);
  int num_blocks = (p->num_wires+255) >> 8; 

  //printf("cnc_gpu(%d, %p, %p, %p, %p, %d)\n",
  // 	 p->first_wire.value, wires.array_base,
  // 	 pvt.array_base, owned.array_base, ghost.array_base, flag);
  calc_new_currents_kernel<<<num_blocks,256>>>(p->first_wire,
                                               p->num_wires,
                                               wires, pvt, owned, ghost,
                                               flag);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__host__
void sanity_check_wires_gpu(CircuitPiece *p,
                            GPU_Accessor wires)
{
  //printf("Wire bounds are %ld and %ld\n",wires.first_elmt,wires.last_elmt);
  int num_blocks = (p->num_wires+255) >> 8;
  sanity_check_positions<<<num_blocks,256>>>(p->first_wire,
                                             p->num_wires,
                                             wires);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
}

template<typename REDOP>
__device__ __forceinline__
void reduce_local(GPU_Accessor pvt, GPU_Reducer owned, GPU_Reducer ghost,
                  PointerLocation loc, ptr_t<CircuitNode> ptr, typename REDOP::RHS value)
{
#ifndef ALL_PRIVATE
  switch (loc)
  {
    case PRIVATE_PTR:
      pvt.template reduce<REDOP,CircuitNode,typename REDOP::RHS>(ptr, value);
      break;
    case SHARED_PTR:
      owned.template reduce<REDOP,CircuitNode,typename REDOP::RHS>(ptr, value);
      break;
    case GHOST_PTR:
      ghost.template reduce<REDOP,CircuitNode,typename REDOP::RHS>(ptr, value);
      break;
    default:
      printf("Bad pointer location %d at pointer %d\n", loc, ptr.value);
      assert(false);
  }
#else
  pvt.template reduce<REDOP,CircuitNode,typename REDOP::RHS>(ptr, value);
#endif
}

__global__
void distribute_charge_kernel(ptr_t<CircuitWire> first,
                              int num_wires,
                              GPU_Accessor wires,
                              GPU_Accessor pvt,
                              GPU_Reducer owned,
                              GPU_Reducer ghost,
                              int flag)
{
#ifndef DISABLE_MATH
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_wires)
  {
    ptr_t<CircuitWire> local_ptr;
    local_ptr.value = first.value + tid;

    CircuitWire &wire = wires.ref(local_ptr);
    //if(//((local_ptr.value >= 300220) && (local_ptr.value <= 300229)) ||
    //   //((local_ptr.value >= 299710) && (local_ptr.value <= 299720)) ||
    //   (wire.in_loc < 0) || (wire.in_loc > 2) ||
    //   (wire.out_loc < 0) || (wire.out_loc > 2) ||
    //   (wire.in_ptr.value < ((local_ptr.value/10000)*2500)) ||
    //   (wire.in_ptr.value >= (((local_ptr.value/10000)+1)*2500)))
    //  printf("wire[%d] = %d(%d) -> %d(%d)\n",
    //	     local_ptr.value, wire.in_ptr.value, wire.in_loc,
    //	     wire.out_ptr.value, wire.out_loc);

    float dt = DELTAT;

    //if(wire.in_ptr.value == 9999)
    //  printf("in_loc[9999] = %d\n", wire.in_loc);
    //if(wire.out_ptr.value == 9999)
    //  printf("out_loc[9999] = %d\n", wire.out_loc);
    //if(wire.in_ptr.value == 8940) {
      //printf("w[%d] n[%d/%d]\n", local_ptr.value, wire.in_ptr.value, wire.in_loc);
    reduce_local<GPUAccumulateCharge>(pvt, owned, ghost, wire.in_loc, wire.in_ptr, -dt * wire.current[0]);
    //}
    reduce_local<GPUAccumulateCharge>(pvt, owned, ghost, wire.out_loc, wire.out_ptr, dt * wire.current[WIRE_SEGMENTS-1]);
  }
#endif
}

__host__
void distribute_charge_gpu(CircuitPiece *p,
                           GPU_Accessor wires,
                           GPU_Accessor pvt,
                           GPU_Reducer owned,
                           GPU_Reducer ghost,
                           int flag)
{
  RegionRuntime::LowLevel::DetailedTimer::ScopedPush sp(TIME_SYSTEM+2);
  int num_blocks = (p->num_wires+255) >> 8;
  distribute_charge_kernel<<<num_blocks,256>>>(p->first_wire,
                                               p->num_wires,
                                               wires, pvt, owned, ghost,
                                               flag);

  CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__global__
void update_voltages_kernel(ptr_t<CircuitNode> first,
                            int num_nodes,
                            GPU_Accessor pvt,
                            GPU_Accessor owned,
                            GPU_Accessor locator,
                            int flag)
{
#ifndef DISABLE_MATH
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_nodes)
  {
    ptr_t<int> locator_ptr;
    locator_ptr.value = first.value + tid;
    ptr_t<CircuitNode> local_node;
    local_node.value = first.value + tid;
    // Figure out if this node is pvt or not
    {
      int is_pvt = locator.read(locator_ptr);
      //if(locator_ptr.value == 9999) printf("pvt[9999] = %d\n", is_pvt);
#ifndef ALL_PRIVATE
      if (is_pvt)
#endif
      {
        CircuitNode &cur_node = pvt.ref(local_node);
        // charge adds in, and then some leaks away
        cur_node.voltage += cur_node.charge / cur_node.capacitance;
        cur_node.voltage *= (1 - cur_node.leakage);
        cur_node.charge = 0;
      }
#ifndef ALL_PRIVATE
      else
      {
        CircuitNode &cur_node = owned.ref(local_node);
        // charge adds in, and then some leaks away
        cur_node.voltage += cur_node.charge / cur_node.capacitance;
        cur_node.voltage *= (1 - cur_node.leakage);
        cur_node.charge = 0;
      }
#endif
    }
  }
#endif
}

__host__
void update_voltages_gpu(CircuitPiece *p,
                         GPU_Accessor pvt,
                         GPU_Accessor owned,
                         GPU_Accessor locator,
                         int flag)
{
  RegionRuntime::LowLevel::DetailedTimer::ScopedPush sp(TIME_SYSTEM+3);
  int num_blocks = (p->num_nodes+255) >> 8;

  update_voltages_kernel<<<num_blocks,256>>>(p->first_node,
                                             p->num_nodes,
                                             pvt, owned, locator,
                                             flag);

  CUDA_SAFE_CALL(hipDeviceSynchronize());
}

